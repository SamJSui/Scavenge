#include "particle.cuh"

SCAVENGE_NAMESPACE_BEGIN

/* Initialize the static random number generators for the float interval [-1.0, 1.0 ] */

std::default_random_engine Particle::gen_ = std::default_random_engine(time(0));
std::uniform_real_distribution<float> Particle::dist_ = std::uniform_real_distribution<float>(-1.0, 1.0);

/* Default values for static world values for the simulation */

float Particle::world_width_ = 100.0f;
float Particle::world_height_ = 100.0f;

Particle::Particle() {
  pos_ = vec2(dist_(gen_) * world_width_, dist_(gen_) * world_height_);
  best_pos_ = pos_;
  vel_ = vec2(0.0f, 0.0f);
}

SCAVENGE_NAMESPACE_END