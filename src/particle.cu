#include "particle.h"

std::default_random_engine Particle::gen_ = std::default_random_engine(time(0));
std::uniform_real_distribution<float> Particle::dist_ = std::uniform_real_distribution<float>(-1.0, 1.0);

Particle::Particle() {
  pos_ = vec2(dist_(gen_) * 100.0f, dist_(gen_) * 100.0f);
  best_pos_ = pos_;
  vel_ = vec2(0.0f, 0.0f);
}
