#include "pso.cuh"

SCAVENGE_NAMESPACE_BEGIN

/* PSO Core Class Definitions */

PSO::PSO(
  unsigned int num_particles, // aggregate init
  float inertia, 
  float cognition, 
  float social) 
  : num_particles_(num_particles),
    inertia_(inertia),
    cognition_(cognition),
    social_(social) {

  /* Allocate particles */

  particles_ = new Particle[num_particles_];
  dev_particles_ = nullptr;

  /* Test Function Pointers */

  test_fn_ = nullptr;
  dev_test_fn_ = nullptr;

  /* Simulation Global Best */

  best_idx_ = 0;
  best_fitness_ = 0.0f;
  
  /* CUDA */

  if (globals::device_count > 0) 
    device_init();
  else 
    settings_.use_gpu_ = false;
}

PSO::~PSO() {
  if (particles_)
    free(particles_);
  if (dev_particles_)
    hipFree(dev_particles_);
}

Particle& PSO::operator[](const unsigned int& idx) {
  return particles_[idx];
}

void PSO::run(const unsigned int epochs) {
  settings_.epochs_ = epochs;
  if (test_fn_ == nullptr) {
    std::string NO_TEST_FUNCTION = 
      "ERROR: PSO run() called without a test function";
    std::cerr << NO_TEST_FUNCTION << std::endl;
    std::exit(EXIT_FAILURE);
  }
  if (globals::device_count && settings_.use_gpu_)
    simulate_gpu();
  else
    simulate_cpu();
}

/* PSO Setters */

void PSO::set_gpu(const bool& gpu) {
  settings_.use_gpu_ = gpu;
}

void PSO::set_test_function(const ScavengeTestFunction test_fn) {
  test_fn_ = test_fn;
}

/* CPU Simulation */

void PSO::simulate_cpu() {
  unsigned int iteration = 0;
  do {
    
    iteration++;
  } while(iteration < settings_.epochs_);
}

void PSO::update_particle_position() {
  for (unsigned int idx = 0; idx < num_particles_; idx++) {
    ;
  }
}

void PSO::update_particle_velocity() {
  for (unsigned int idx = 0; idx < num_particles_; idx++) {
    ;
  }
}

void PSO::update_global_best() {
  for (unsigned int idx = 0; idx < num_particles_; idx++) {
    ;
  }
}

void PSO::simulate_gpu() {
  ;
}

void PSO::device_init() {
  unsigned int grid_size;
  unsigned int particles_bytes;
  hiprandState *state;

  settings_.use_gpu_ = true;
  particles_bytes = sizeof(Particle) * num_particles_;

  hipMalloc((void **) &dev_particles_, particles_bytes);
  hipMalloc(&state, sizeof(hiprandState) * globals::block_size);
  hipMemcpy(dev_particles_, particles_, particles_bytes, hipMemcpyHostToDevice);

  grid_size = (num_particles_ + globals::block_size - 1) / globals::block_size;
  kernel_curand_setup<<<grid_size, globals::block_size>>>(state, rand() % INT_MAX);
}

/* PSO Global (CUDA Kernel) / Device Helper Functions */

namespace pso {

}

SCAVENGE_NAMESPACE_END