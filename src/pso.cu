#include "pso.cuh"

SCAVENGE_NAMESPACE_BEGIN

/* PSO Core Class Definitions */

PSO::PSO(
  unsigned int num_particles, // aggregate init
  float inertia, 
  float cognition, 
  float social) 
  : num_particles_(num_particles),
    inertia_(inertia),
    cognition_(cognition),
    social_(social) {
  particles_ = new Particle[num_particles_]; // particles
  dev_particles_ = nullptr;

  best_idx_ = 0; // global best
  best_fitness_ = 0.0f;
  
  if (globals::device_count > 0) 
    device_init();
  else 
    settings_.use_gpu_ = false;
}

PSO::~PSO() {
  if (particles_)
    free(particles_);
  if (dev_particles_)
    hipFree(dev_particles_);
}

Particle& PSO::operator[](const unsigned int& idx) {
  return particles_[idx];
}

void PSO::run(const unsigned int epochs) {
  settings_.epochs_ = epochs;
  if (globals::device_count && settings_.use_gpu_)
    simulate_gpu();
  else
    simulate_cpu();
}

void PSO::set_gpu(const bool& gpu) {
  settings_.use_gpu_ = gpu;
}

void PSO::simulate_cpu() {
  unsigned int iteration = 0;
  do {
    
    iteration++;
  } while(iteration < settings_.epochs_);
}

void PSO::simulate_gpu() {
  ;
}

void PSO::device_init() {
  unsigned int grid_size;
  unsigned int particles_bytes;
  hiprandState *state;

  settings_.use_gpu_ = true;
  particles_bytes = sizeof(Particle) * num_particles_;

  hipMalloc((void **) &dev_particles_, particles_bytes);
  hipMalloc(&state, sizeof(hiprandState) * globals::block_size);
  hipMemcpy(dev_particles_, particles_, particles_bytes, hipMemcpyHostToDevice);

  grid_size = (num_particles_ + globals::block_size - 1) / globals::block_size;
  kernel_curand_setup<<<grid_size, globals::block_size>>>(state, rand() % INT_MAX);
}

/* PSO Global (CUDA Kernel) / Device Helper Functions */

namespace pso {

  void update_particle_position() {
    ;
  }

  void update_particle_velocity() {
    ;
  }

  void update_global_best() {
    ;
  }

}

SCAVENGE_NAMESPACE_END