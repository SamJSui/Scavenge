#include "hip/hip_runtime.h"
#include "pso.cuh"

SCAVENGE_NAMESPACE_BEGIN

/* PSO Global (CUDA Kernel) / Device Helper Functions */

namespace pso {

  __global__
  void update_particle_position_gpu() {
    vec2 a(1.0, 2.0);
  }
  __global__
  void update_particle_velocity_gpu() {
    ;
  }
  __global__
  void update_global_best_gpu() {
    ;
  }

}

/* PSO Core Class Definitions */

PSO::PSO(
  unsigned int n_p, // aggregate init
  float i, 
  float c, 
  float s) 
  : num_particles_(n_p),
    inertia(i),
    cognition(c),
    social(s) {

  /* Allocate particles */

  particles_ = new Particle[num_particles_];
  dev_particles_ = nullptr;

  /* Simulation Global Best */

  best_idx_ = 0;
  
  /* CUDA */

  if (globals::device_count > 0) 
    device_init();
  else 
    settings_.use_gpu_ = false;
}

PSO::~PSO() {
  if (particles_)
    free(particles_);
  if (dev_particles_)
    hipFree(dev_particles_);
}

Particle& PSO::operator[](const unsigned int& idx) {
  return particles_[idx];
}

void PSO::run(const unsigned int epochs) {
  settings_.epochs_ = epochs;
  best_fitness_ = test_fn(particles_[0].pos_);
  printf("%f\n", best_fitness_);
  if (globals::device_count && settings_.use_gpu_)
    simulate_gpu();
  else
    simulate_cpu();
}

/* PSO Setters */

void PSO::set_gpu(const bool& gpu) {
  settings_.use_gpu_ = gpu;
}

/* CPU Simulation */

void PSO::simulate_cpu() {
  unsigned int iteration = 0;
  do {
    
    iteration++;
  } while(iteration < settings_.epochs_);
}

void PSO::update_particle_position() {
  for (unsigned int idx = 0; idx < num_particles_; idx++) {
    float new_fitness = test_fn(particles_[idx].pos_);
    if (new_fitness < particles_[idx].best_fitness_)
      particles_[idx].best_fitness_ = new_fitness;
  }
}

void PSO::update_particle_velocity() {
  for (unsigned int idx = 0; idx < num_particles_; idx++) {
    float rand_1 = Particle::dist_(Particle::gen_);
    float rand_2 = Particle::dist_(Particle::gen_);
    vec2 global_pos = particles_[best_idx_].best_pos_;

    particles_[idx].vel_.x = inertia * particles_[idx].vel_.x + // Calculate particle's velocity
      cognition * rand_1 * (particles_[idx].best_pos_.x - particles_[idx].best_pos_.x) + 
      social * rand_2 * (global_pos.x - particles_[idx].pos_.x);

    particles_[idx].vel_.y = inertia * particles_[idx].vel_.y +
      cognition * rand_1 * (particles_[idx].best_pos_.y - particles_[idx].best_pos_.y) + 
      social * rand_2 * (global_pos.y - particles_[idx].pos_.y);
  }
}

void PSO::update_global_best() {
  for (unsigned int idx = 0; idx < num_particles_; idx++) {
    
  }
}

void PSO::simulate_gpu() {
  unsigned int grid_size = (num_particles_ + globals::block_size - 1) / globals::block_size;
  unsigned int iteration = 0;
  do {
    pso::update_particle_position_gpu<<<grid_size, globals::block_size>>>();
    iteration++;
  } while(iteration < settings_.epochs_);
}

void PSO::device_init() {
  unsigned int grid_size;
  unsigned int particles_bytes;
  hiprandState *state;

  settings_.use_gpu_ = true;
  particles_bytes = sizeof(Particle) * num_particles_;

  hipMalloc((void **) &dev_particles_, particles_bytes);
  hipMalloc(&state, sizeof(hiprandState) * globals::block_size);
  hipMemcpy(dev_particles_, particles_, particles_bytes, hipMemcpyHostToDevice);

  grid_size = (num_particles_ + globals::block_size - 1) / globals::block_size;
  kernel_curand_setup<<<grid_size, globals::block_size>>>(state, rand() % INT_MAX);
}

SCAVENGE_NAMESPACE_END