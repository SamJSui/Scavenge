#include "pso.cuh"

SCAVENGE_NAMESPACE_BEGIN

PSO::PSO(
  unsigned int num_particles, // aggregate init
  float inertia, 
  float cognition, 
  float social) 
  : num_particles_(num_particles),
    inertia_(inertia),
    cognition_(cognition),
    social_(social) {
  particles_ = new Particle[num_particles_]; // particles
  d_particles_ = nullptr;

  best_idx_ = 0; // global best
  best_fitness_ = 0.0f;
  
  if (globals::device_count > 0) 
    device_init();
  else 
    settings_.use_gpu_ = false;
}

PSO::~PSO() {
  if (particles_)
    free(particles_);
  if (d_particles_)
    hipFree(d_particles_);
}

Particle PSO::operator[](unsigned int idx) {
  return particles_[idx];
}

void PSO::run(unsigned int epochs) {
  settings_.epochs_ = epochs;
  if (globals::device_count && settings_.use_gpu_);
  else;
}

void PSO::set_gpu(bool gpu) {
  settings_.use_gpu_ = gpu;
}

void PSO::device_init() {
  unsigned int grid_size;
  ssize_t particles_bytes;
  hiprandState *state;

  settings_.use_gpu_ = true;
  particles_bytes = sizeof(Particle) * num_particles_;

  hipMalloc((void **) &d_particles_, particles_bytes);
  hipMalloc(&state, sizeof(hiprandState) * globals::block_size);
  hipMemcpy(d_particles_, particles_, particles_bytes, hipMemcpyHostToDevice);

  grid_size = (num_particles_ + globals::block_size - 1) / globals::block_size;
  kernel_curand_setup<<<grid_size, globals::block_size>>>(state, rand() % INT_MAX);
}

SCAVENGE_NAMESPACE_END