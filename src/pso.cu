#include "pso.h"

/// pso/class.cuh ///

PSO::PSO(
    unsigned int num_particles, // aggregate init
    float inertia, 
    float cognition, 
    float social) 
    : num_particles_(num_particles),
      inertia_(inertia),
      cognition_(cognition),
      social_(social) {
  particles_ = new Particle[num_particles_]; // particles
  d_particles_ = nullptr;
  best_idx_ = 0; // global best
  best_fitness_ = 0.0f;
  hipGetDeviceCount(&gpu_device_count_); // cuda
  if (hipGetDeviceCount > 0) 
    device_init();
  else 
    use_gpu_ = 0;
}

PSO::~PSO() {
  if (particles_)
    free(particles_);
  if (d_particles_)
    hipFree(d_particles_);
}

Particle PSO::operator[](unsigned int idx) {
  return particles_[idx];
}

void PSO::run(unsigned int epochs) {
  if (gpu_device_count_ && use_gpu_)
    simulate_gpu();
  else
    simulate_cpu();
}

void PSO::device_init() {
  unsigned int grid_size;
  ssize_t particles_bytes;
  hiprandState *state;
  use_gpu_ = 1;
  particles_bytes = sizeof(Particle) * num_particles_;
  hipMalloc((void **) &d_particles_, particles_bytes);
  hipMalloc(&state, sizeof(hiprandState) * config::block_size);
  hipMemcpy(d_particles_, particles_, particles_bytes, hipMemcpyHostToDevice);
  printf("host x: %f\n", particles_[39].best_pos_.x);
  grid_size = (num_particles_ + config::block_size - 1) / config::block_size;
  g_kernel_setup<<<grid_size, config::block_size>>>(state, rand() % INT_MAX, num_particles_);
  g_pso_simulate<<<grid_size, config::block_size>>>(d_particles_, num_particles_);
}

void PSO::simulate_cpu() {

}

void PSO::simulate_gpu() {

}

//// pso/simulate.cuh ////

CUDA_GLOBAL void g_kernel_setup(
    hiprandState *state, 
    unsigned int seed, 
    unsigned int num_particles) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= num_particles) return;
  hiprand_init(seed, idx, 0, &state[idx]);
}

CUDA_GLOBAL void g_pso_simulate(Particle *d_particles, unsigned int num_particles) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= num_particles) return;
  if (idx == 0) {
    printf("dev x: %f\n", d_particles[39].best_pos_.x);
  }
}