#include "config.cuh"

SCAVENGE_NAMESPACE_BEGIN

int globals::device_count = 0;
unsigned int globals::block_size = 256;

Config::Config() {
  hipGetDeviceCount(&globals::device_count);
  use_gpu_ = globals::device_count;
}

SCAVENGE_NAMESPACE_END