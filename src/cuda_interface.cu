#include "hip/hip_runtime.h"
#include "cuda_interface.cuh"

__global__ 
void kernel_curand_setup(hiprandState *state, unsigned int seed) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}