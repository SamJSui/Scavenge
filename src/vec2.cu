#include "vec2.cuh"

SCAVENGE_NAMESPACE_BEGIN

__host__ __device__ 
vec2::vec2()
  : x(0.0f),
    y(0.0f) {}

__host__ __device__
vec2::vec2(const float a, const float b)
  : x(a),
    y(b) {}

__host__ __device__
void vec2::operator=(const vec2& vec2_operand) {
  x = vec2_operand.x;
  y = vec2_operand.y;
}

__host__ __device__
void vec2::operator+=(const vec2& vec2_operand) {
  x += vec2_operand.x;
  y += vec2_operand.y;
}

__host__ __device__
void vec2::operator-=(const vec2& vec2_operand) {
  x -= vec2_operand.x;
  y -= vec2_operand.y;
}

__host__ __device__
vec2 vec2::operator+(const vec2& vec2_operand) {
  return vec2(x + vec2_operand.x, y + vec2_operand.y);
}


__host__ __device__
vec2 vec2::operator-(const vec2& vec2_operand) {
  return vec2(x - vec2_operand.x, y - vec2_operand.y);
}

SCAVENGE_NAMESPACE_END