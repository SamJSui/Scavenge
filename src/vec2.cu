#include "vec2.h"

CUDA_HOSTDEV vec2::vec2() {
  x = 0.0f;
  y = 0.0f;
}

CUDA_HOSTDEV vec2::vec2(const float a, const float b) {
  x = a;
  y = b;
}

CUDA_HOSTDEV void vec2::operator=(const vec2& p) {
  x = p.x;
  y = p.y;
}

CUDA_HOSTDEV void vec2::operator+=(const vec2& p) {
  x += p.x;
  y += p.y;
}

CUDA_HOSTDEV void vec2::operator-=(const vec2& p) {
  x -= p.x;
  y -= p.y;
}

CUDA_HOSTDEV vec2 vec2::operator+(const vec2& p) {
  return vec2(x + p.x, y + p.y);
}

CUDA_HOSTDEV vec2 vec2::operator-(const vec2& p) {
  return vec2(x - p.x, y - p.y);
}